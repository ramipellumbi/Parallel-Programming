#define FP float
#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

void write_data_to_file(const char *filename,   // name of csv
                        const char *program,    // 'task1', 'task2', etc.
                        const char *multiplier, // 'cpu' or 'gpu'
                        int n,
                        int p,
                        int m,
                        int block_x,
                        int block_y,
                        int grid_x,
                        int grid_y,
                        float exe_time)
{
    // check if the file exists
    FILE *check_file = fopen(filename, "r");
    bool does_file_exist = false;
    if (check_file != NULL)
    {
        fclose(check_file);
        does_file_exist = true;
    }

    // open the file with means to append
    FILE *fp = fopen(filename, "a");
    if (fp == NULL)
    {
        fprintf(stderr, "Could not open or create file: %s\n", filename);
        return;
    }

    // if the file does not exist, add the header row
    if (!does_file_exist)
    {
        fprintf(fp, "program,multiplier,precision,n,p,m,block_x,block_y,grid_x,grid_y,exe_time\n");
    }

    // add data to row
    fprintf(fp, "\"%s\",\"%s\",\"%s\",%d,%d,%d,%d,%d,%d,%d,%f\n", program, multiplier, TOSTRING(FP), n, p, m, block_x, block_y, grid_x, grid_y, exe_time);
    fclose(fp);
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 */
void cpu_matrixmult_rectangular_kij(FP *A, FP *B, FP *C, int n, int p, int m)
{
    FP r;
    // the dot product between a row of A and column of B is between p numbers
    for (size_t k = 0; k < p; k++)
    {
        // there are n rows in A
        for (size_t i = 0; i < n; i++)
        {
            size_t ia = i * p + k; // row i column k of A
            r = A[ia];

            // there are m columns in b
            for (size_t j = 0; j < m; j++)
            {
                size_t ib = k * m + j; // row k column j of B
                size_t ic = i * m + j; // row i column j of C

                C[ic] += r * B[ib];
            }
        }
    }
}

/**
 * Print matrix M (stored row wise)
 */
void print_matrix(FP *M, const char *name, size_t nrow, size_t ncol)
{
    printf("Matrix %s\n", name);
    printf("[\n");
    for (size_t i = 0; i < nrow; i++)
    {
        printf("[");
        for (size_t j = 0; j < ncol; j++)
        {
            printf("%f,", M[i * ncol + j]);
        }
        printf("],\n");
    }
    printf("]\n");
}

int main(int argc, char **argv)
{

    FP *A, *B, *C;
    int n, p, m;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms;

    if (argc != 4)
    {
        printf("Usage: task1 <matrix dim n> <matrix dim p> <matrix dim m>\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);
    printf("%d,%d,%d", n, p, m);

    size_t size_A = n * p * sizeof(FP);
    size_t size_B = p * m * sizeof(FP);
    size_t size_C = n * m * sizeof(FP);

    A = (FP *)malloc(size_A);
    B = (FP *)malloc(size_B);
    C = (FP *)malloc(size_C);

    srand(12345);
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < p; j++)
        {
            A[i * p + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < p; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            B[i * m + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            C[i * m + j] = 0.0;
        }
    }

    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    cpu_matrixmult_rectangular_kij(A, B, C, n, p, m); // do calculation on host
    hipEventRecord(stop, 0);                         // instrument code to measue end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    write_data_to_file("out/task1.csv", "task1a", "cpu", n, p, m, 0, 0, 0, 0, elapsed_time_ms);

    // -------------- clean up ---------------------------------------
    free(A);
    free(B);
    free(C);

    return 0;
}