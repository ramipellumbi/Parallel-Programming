/**
 * This program is a carbon copy of task3.cu - since that was implemented in the 
 * general way from the start. It was easier to think about the problem 
 * generally when referencing Kirk & Kwu + the slides.
*/
#define FP float
#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)
#define NTB 3

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

void write_data_to_file(const char *filename,   // name of csv
                        const char *program,    // 'task1', 'task2', etc.
                        const char *multiplier, // 'cpu' or 'gpu'
                        int n,
                        int p,
                        int m,
                        int block_x,
                        int block_y,
                        int grid_x,
                        int grid_y,
                        float exe_time)
{
    // check if the file exists
    FILE *check_file = fopen(filename, "r");
    bool does_file_exist = false;
    if (check_file != NULL)
    {
        fclose(check_file);
        does_file_exist = true;
    }

    // open the file with means to append
    FILE *fp = fopen(filename, "a");
    if (fp == NULL)
    {
        fprintf(stderr, "Could not open or create file: %s\n", filename);
        return;
    }

    // if the file does not exist, add the header row
    if (!does_file_exist)
    {
        fprintf(fp, "program,multiplier,precision,n,p,m,block_x,block_y,grid_x,grid_y,NTB,exe_time\n");
    }

    // add data to row
    fprintf(fp, "\"%s\",\"%s\",\"%s\",%d,%d,%d,%d,%d,%d,%d,%d,%f\n", program, multiplier, TOSTRING(FP), n, p, m, block_x, block_y, grid_x, grid_y, NTB, exe_time);
    fclose(fp);
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 */
void cpu_matrixmult_rectangular_kij(FP *A, FP *B, FP *C, int n, int p, int m)
{
    FP r;
    // the dot product between a row of A and column of B is between p numbers
    for (size_t k = 0; k < p; k++)
    {
        // there are n rows in A
        for (size_t i = 0; i < n; i++)
        {
            size_t ia = i * p + k; // row i column k of A
            r = A[ia];

            // there are m columns in b
            for (size_t j = 0; j < m; j++)
            {
                size_t ib = k * m + j; // row k column j of B
                size_t ic = i * m + j; // row i column j of C

                C[ic] += r * B[ib];
            }
        }
    }
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 *
 * Follows psuedo-code from lecture 12 on multi-tiled matrix multiplication kernel
 */
__global__ void gpu_matrixmult_rectangular_shared(FP *A, FP *B, FP *C, int n, int p, int m, int TILE_WIDTH)
{
    // the textbook uses two double arrays - I interpreted not copying
    // the data structure as continuing to use single arrays
    extern __shared__ FP tiles[];
    FP *Ads = &tiles[0]; // TILE_WIDTH xTILE_WIDTH
    FP *Bds[NTB];        // pointer to an array of NTB elements each of size TILE_WIDTH x TILE_WIDTH

    // initialize cvalues
    FP cvalues[NTB];
    for (size_t kt = 0; kt < NTB; kt++)
    {
        // tiles[0] to tiles[TILE_WIDTH*TILE_WIDTH - 1] is occupied by Ads
        // since Bds is NTB x TILE_WIDTH x TILE_WIDTH, offset each Bds[i] by (kt+1)*TILE_WIDTH*TILE_WIDTH in tiles
        Bds[kt] = &tiles[(kt + 1) * TILE_WIDTH * TILE_WIDTH];
        cvalues[kt] = 0.;
    }

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    // col needs to be dynamically computed based on NTB idx

    int tile_idx = ty * TILE_WIDTH + tx;

    // Loop over the A and B tiles required to compute the C element
    for (size_t ph = 0; ph < ceil((double)p / (double)TILE_WIDTH); ph++)
    {
        // collaborative loading of A and B tiles into shared memory

        int col_bound_A = ph * TILE_WIDTH + tx;
        int row_bound_B = ph * TILE_WIDTH + ty;

        // load Ads
        if (row < n && col_bound_A < p)
        {
            int indexa = row * p + col_bound_A;
            Ads[tile_idx] = A[indexa];
        } else {
            Ads[tile_idx] = 0.;
        }

        // load multiple tiles of B into the Bds array
        for (size_t kt = 0; kt < NTB; kt++)
        {
            int col_offset = tx + NTB * blockDim.x * bx + kt * TILE_WIDTH;
            int indexb = row_bound_B * m + col_offset;

            if (col_offset < m && row_bound_B < p)
            {
                Bds[kt][tile_idx] = B[indexb];
            } else {
                Bds[kt][tile_idx] = 0;
            }
        }
        __syncthreads();

        for (size_t k = 0; k < TILE_WIDTH; k++)
        {
            for (size_t kt = 0; kt < NTB; kt++)
            {
                // row ty column k of Ads with row k column tx of Bds[i]
                cvalues[kt] += Ads[ty * TILE_WIDTH + k] * Bds[kt][k * TILE_WIDTH + tx];
            }
        }
        __syncthreads();
    }

    for (size_t kt = 0; kt < NTB; kt++)
    {
        int col_offset = tx + NTB * blockDim.x * bx + kt * TILE_WIDTH;
        if (col_offset < m && row < n)
        {
            C[row * m + col_offset] = cvalues[kt];
        }
    }
}

int main(int argc, char **argv)
{

    FP *A, *B, *C; // matrices on host
    FP *HOST_C;
    FP *dev_A, *dev_B, *dev_C; // matrices on device
    int n, p, m;               // matrix dimensions

    int gpucount = 0;   // Count of available GPUs
    int gpunum = 0;     // Device number to use
    int Grid_Dim_x = 1; // Grid dimension, x and y
    int Grid_Dim_y = 1;
    int Block_Dim_x = 1; // Block dimension, x and y
    int Block_Dim_y = 1;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms;   // which is applicable for asynchronous code also
    hipError_t errorcode;

    // --------------------SET PARAMETERS AND DATA -----------------------
    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice)
    {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else
    {
        printf("Device count = %d\n", gpucount);
    }

    if (argc != 8)
    {
        printf("Usage: task1 <matrix dim n> <matrix dim p> <matrix dim m> <block dim x> <block dim y> <grid dim x> <grid dim y>\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);

    Block_Dim_x = atoi(argv[4]); // Rectangular block
    Block_Dim_y = atoi(argv[5]);
    int TILE_WIDTH = Block_Dim_x;
    if (Block_Dim_x * Block_Dim_y > 1024)
    {
        printf("Error, too many threads in block\n");
        exit(-1);
    }

    Grid_Dim_x = ((atoi(argv[6]) - 1) / NTB) + 1; // bash file computes (m -1) / blockx + 1 and we want (m - 1) / blockx * NTB + 1
    Grid_Dim_y = atoi(argv[7]);
    if (Grid_Dim_x * NTB * Block_Dim_x < m)
    {
        printf("%d,%d\n", Grid_Dim_x, Block_Dim_x);
        printf("Error, number of threads in x dimensions less than number of array elements\n");
        exit(-1);
    }
    if (Grid_Dim_y * Block_Dim_y < n)
    {
        printf("Error, number of threads in y dimensions less than number of array elements\n");
        exit(-1);
    }

    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    printf("Matrix Dimension = %d\n", n);
    printf("Block_Dim_x = %d, Block_Dim_y = %d, Grid_Dim_x = %d, Grid_Dim_y = %d\n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

    dim3 Grid(Grid_Dim_x, Grid_Dim_y);    // Grid structure
    dim3 Block(Block_Dim_x, Block_Dim_y); // Block structure

    size_t size_A = n * p * sizeof(FP);
    size_t size_B = p * m * sizeof(FP);
    size_t size_C = n * m * sizeof(FP);

    A = (FP *)malloc(size_A); // dynamically allocated memory for arrays on host
    B = (FP *)malloc(size_B); // dynamically allocated memory for arrays on host
    C = (FP *)malloc(size_C); // results from GPU
    HOST_C = (FP *)malloc(size_C);

    srand(12345);
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < p; j++)
        {
            A[i * p + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < p; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            B[i * m + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            C[i * m + j] = 0.;
            // HOST_C[i * m + j] = 0.;
        }
    }

    // ------------- COMPUTATION DONE ON GPU ----------------------------
    hipMalloc((void **)&dev_A, size_A); // allocate memory on device
    hipMalloc((void **)&dev_B, size_B);
    hipMalloc((void **)&dev_C, size_C);
    hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    size_t TW = (NTB + 1) * TILE_WIDTH * TILE_WIDTH * sizeof(FP); // Bds needs NTB * TILE_WIDTH * TILE_WIDTH and Ads needs TILE_WIDTH * TILE_WIDTH FP's
    gpu_matrixmult_rectangular_shared<<<Grid, Block, TW>>>(dev_A, dev_B, dev_C, n, p, m, TILE_WIDTH);

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost);

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
    write_data_to_file("out/task4.csv", "task4", "gpu", n, p, m, Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y, elapsed_time_ms);

    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

    hipEventRecord(start, 0); // use same timing

    cpu_matrixmult_rectangular_kij(A, B, HOST_C, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

    hipEventRecord(stop, 0); // instrument code to measue end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); 

    // ------------------- check device creates correct results -----------------

    double error, suma, sumb, sumc, ai, bi, ci;
    suma = 0.;
    sumb = 0;
    sumc = 0;
    for (size_t i = 0; i < n * p; i++)
    {
        ai = (double)A[i];
        suma += ai * ai;
    }
    for (size_t i = 0; i < p * m; i++)
    {
        bi = (double)B[i];
        sumb += bi * bi;
    }
    for (size_t i = 0; i < n * m; i++)
    {
        ci = (double)C[i] - (double)HOST_C[i];
        sumc += ci * ci;
    }
    suma = sqrt(suma);
    sumb = sqrt(sumb);
    sumc = sqrt(sumc);
    error = sumc / (suma * sumb);
    printf("Approximate relative error between GPU and CPU: %e\n", error);

    // -------------- clean up ---------------------------------------
    free(A);
    free(B);
    free(C);
    free(HOST_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}