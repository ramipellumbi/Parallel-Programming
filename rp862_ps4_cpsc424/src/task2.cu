#define FP double
#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

void write_data_to_file(const char *filename,   // name of csv
                        const char *program,    // 'task1', 'task2', etc.
                        const char *multiplier, // 'cpu' or 'gpu'
                        int n,
                        int p,
                        int m,
                        int block_x,
                        int block_y,
                        int grid_x,
                        int grid_y,
                        float exe_time)
{
    // check if the file exists
    FILE *check_file = fopen(filename, "r");
    bool does_file_exist = false;
    if (check_file != NULL)
    {
        fclose(check_file);
        does_file_exist = true;
    }

    // open the file with means to append
    FILE *fp = fopen(filename, "a");
    if (fp == NULL)
    {
        fprintf(stderr, "Could not open or create file: %s\n", filename);
        return;
    }

    // if the file does not exist, add the header row
    if (!does_file_exist)
    {
        fprintf(fp, "program,multiplier,precision,n,p,m,block_x,block_y,grid_x,grid_y,exe_time\n");
    }

    // add data to row
    fprintf(fp, "\"%s\",\"%s\",\"%s\",%d,%d,%d,%d,%d,%d,%d,%f\n", program, multiplier, TOSTRING(FP), n, p, m, block_x, block_y, grid_x, grid_y, exe_time);
    fclose(fp);
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 *
 * This function was modified and extensively based on Figure 4.16
 * and Figure 4.20 along with section 4.7 in Chapter 4 of Kirk and Hwu's
 * `Programming Massively Parallel Processors`
 */
__global__ void gpu_matrixmult_rectangular_shared(FP *A, FP *B, FP *C, int n, int p, int m, int TILE_WIDTH)
{
    // the textbook uses two double arrays - I interpreted not copying
    // the data structure as continuing to use single arrays
    extern __shared__ FP tiles[];
    FP *Ads = &tiles[0];
    FP *Bds = &tiles[TILE_WIDTH * TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int tile_idx = ty * TILE_WIDTH + tx;

    FP Cvalue = 0;
    // Loop over the A and B tiles required to compute the C element
    for (size_t ph = 0; ph < ceil((double)p / (double)TILE_WIDTH); ph++)
    {
        // collaborative loading of A and B tiles into shared memory
        int col_bound_A = ph * TILE_WIDTH + tx;
        int row_bound_B = ph * TILE_WIDTH + ty;

        int indexa = row * p + col_bound_A;
        int indexb = row_bound_B * m + col;
        if (row < n && col_bound_A < p)
        {
            Ads[tile_idx] = A[indexa];
        }
        if (col < m && row_bound_B < p)
        {
            Bds[tile_idx] = B[indexb];
        }
        __syncthreads();

        for (size_t k = 0; k < TILE_WIDTH; k++)
        {
            // row ty column k of A with row k column tx of B
            Cvalue += Ads[ty * TILE_WIDTH + k] * Bds[k * TILE_WIDTH + tx];
        }
        __syncthreads();
    }
    if (col < m && row < n)
    {
        C[row * m + col] = Cvalue;
    }
}

// void cpu_matrixmult_rectangular_kij(FP *A, FP *B, FP *C, int n, int p, int m)
// {
//     FP r;
//     // the dot product between a row of A and column of B is between p numbers
//     for (size_t k = 0; k < p; k++)
//     {
//         // there are n rows in A
//         for (size_t i = 0; i < n; i++)
//         {
//             size_t ia = i * p + k; // row i column k of A
//             r = A[ia];

//             // there are m columns in b
//             for (size_t j = 0; j < m; j++)
//             {
//                 size_t ib = k * m + j; // row k column j of B
//                 size_t ic = i * m + j; // row i column j of C

//                 C[ic] += r * B[ib];
//             }
//         }
//     }
// }

int main(int argc, char **argv)
{

    FP *A, *B, *C; // matrices on host
    FP *HOST_C;    
    FP *dev_A, *dev_B, *dev_C; // matrices on device
    int n, p, m;               // matrix dimensions

    int gpucount = 0;   // Count of available GPUs
    int gpunum = 0;     // Device number to use
    int Grid_Dim_x = 1; // Grid dimension, x and y
    int Grid_Dim_y = 1;
    int Block_Dim_x = 1; // Block dimension, x and y
    int Block_Dim_y = 1;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms;   // which is applicable for asynchronous code also
    hipError_t errorcode;

    // --------------------SET PARAMETERS AND DATA -----------------------
    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice)
    {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else
    {
        printf("Device count = %d\n", gpucount);
    }

    if (argc != 8)
    {
        printf("Usage: task1 <matrix dim n> <matrix dim p> <matrix dim m> <block dim x> <block dim y> <grid dim x> <grid dim y>\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);

    Block_Dim_x = atoi(argv[4]); // Rectangular block
    Block_Dim_y = atoi(argv[5]);
    int TILE_WIDTH = Block_Dim_x;
    if (Block_Dim_x * Block_Dim_y > 1024)
    {
        printf("Error, too many threads in block\n");
        exit(-1);
    }

    Grid_Dim_x = atoi(argv[6]);
    Grid_Dim_y = atoi(argv[7]);
    if (Grid_Dim_x * Block_Dim_x < m)
    {
        printf("%d,%d", Grid_Dim_x, Block_Dim_x);
        printf("Error, number of threads in x dimensions less than number of array elements\n");
        exit(-1);
    }
    if (Grid_Dim_y * Block_Dim_y < n)
    {
        printf("Error, number of threads in y dimensions less than number of array elements\n");
        exit(-1);
    }

    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    printf("Matrix Dimension = %d\n", n);
    printf("Block_Dim_x = %d, Block_Dim_y = %d, Grid_Dim_x = %d, Grid_Dim_y = %d\n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

    dim3 Grid(Grid_Dim_x, Grid_Dim_y);    // Grid structure
    dim3 Block(Block_Dim_x, Block_Dim_y); // Block structure

    size_t size_A = n * p * sizeof(FP);
    size_t size_B = p * m * sizeof(FP);
    size_t size_C = n * m * sizeof(FP);

    A = (FP *)malloc(size_A); // dynamically allocated memory for arrays on host
    B = (FP *)malloc(size_B); // dynamically allocated memory for arrays on host
    C = (FP *)malloc(size_C); // results from GPU
    // HOST_C = (FP *)malloc(size_C);

    srand(12345);
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < p; j++)
        {
            A[i * p + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < p; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            B[i * m + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            C[i * m + j] = 0.;
            // HOST_C[i * m + j] = 0.;
        }
    }

    // ------------- COMPUTATION DONE ON GPU ----------------------------
    hipMalloc((void **)&dev_A, size_A); // allocate memory on device
    hipMalloc((void **)&dev_B, size_B);
    hipMalloc((void **)&dev_C, size_C);
    hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    size_t TW = 2 * TILE_WIDTH * TILE_WIDTH * sizeof(FP);
    gpu_matrixmult_rectangular_shared<<<Grid, Block, TW>>>(dev_A, dev_B, dev_C, n, p, m, TILE_WIDTH);

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost);

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
    write_data_to_file("out/task2.csv", "task2", "gpu", n, p, m, Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y, elapsed_time_ms);

    // // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

    // cudaEventRecord(start, 0); // use same timing
    // // cudaEventSynchronize(start); // not needed

    // cpu_matrixmult_rectangular_kij(A, B, HOST_C, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

    // cudaEventRecord(stop, 0); // instrument code to measue end time
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&elapsed_time_ms, start, stop);

    // printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

    // // ------------------- check device creates correct results -----------------

    // double error, suma, sumb, sumc, ai, bi, ci;
    // suma = 0.;
    // sumb = 0;
    // sumc = 0;
    // for (size_t i = 0; i < n * p; i++)
    // {
    //     ai = (double)A[i];
    //     suma += ai * ai;
    // }
    // for (size_t i = 0; i < p * m; i++)
    // {
    //     bi = (double)B[i];
    //     sumb += bi * bi;
    // }
    // for (size_t i = 0; i < n * m; i++)
    // {
    //     ci = (double)C[i] - (double)HOST_C[i];
    //     sumc += ci * ci;
    // }
    // suma = sqrt(suma);
    // sumb = sqrt(sumb);
    // sumc = sqrt(sumc);
    // error = sumc / (suma * sumb);
    // printf("Approximate relative error between GPU and CPU: %e\n", error);
    // END OF OPTIONAL SECTION THAT CAN BE OMITTED
    // -------------- clean up ---------------------------------------

    // -------------- clean up ---------------------------------------
    free(A);
    free(B);
    free(C);
    // free(HOST_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}