#define FP float
#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

void write_data_to_file(const char *filename,   // name of csv
                        const char *program,    // 'task1', 'task2', etc.
                        const char *multiplier, // 'cpu' or 'gpu'
                        int n,
                        int p,
                        int m,
                        int block_x,
                        int block_y,
                        int grid_x,
                        int grid_y,
                        float exe_time)
{
    // check if the file exists
    FILE *check_file = fopen(filename, "r");
    bool does_file_exist = false;
    if (check_file != NULL)
    {
        fclose(check_file);
        does_file_exist = true;
    }

    // open the file with means to append
    FILE *fp = fopen(filename, "a");
    if (fp == NULL)
    {
        fprintf(stderr, "Could not open or create file: %s\n", filename);
        return;
    }

    // if the file does not exist, add the header row
    if (!does_file_exist)
    {
        fprintf(fp, "program,multiplier,precision,n,p,m,block_x,block_y,grid_x,grid_y,exe_time\n");
    }

    // add data to row
    fprintf(fp, "\"%s\",\"%s\",\"%s\",%d,%d,%d,%d,%d,%d,%d,%f\n", program, multiplier, TOSTRING(FP), n, p, m, block_x, block_y, grid_x, grid_y, exe_time);
    fclose(fp);
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 */
__global__ void gpu_matrixmult_rectangular(FP *A, FP *B, FP *C, int n, int p, int m)
{

    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    int indexb = col;
    int index = row * m + col;

    if (col < m && row < n)
    {
        int start_index_a = row * p;
        int max_index_a = row * p + p;
        for (int indexa = start_index_a; indexa < max_index_a; indexa++)
        {
            C[index] += A[indexa] * B[indexb];
            indexb += m;
        }
    }
}

/**
 * @param A is n x p - STORED ROW-WISE
 * @param B is p x m - STORED ROW-WISE
 * @param C is n x m - STORED ROW-WISE
 */
void cpu_matrixmult_rectangular_kij(FP *A, FP *B, FP *C, int n, int p, int m)
{
    FP r;
    // the dot product between a row of A and column of B is between p numbers
    for (size_t k = 0; k < p; k++)
    {
        // there are n rows in A
        for (size_t i = 0; i < n; i++)
        {
            size_t ia = i * p + k; // row i column k of A
            r = A[ia];

            // there are m columns in b
            for (size_t j = 0; j < m; j++)
            {
                size_t ib = k * m + j; // row k column j of B
                size_t ic = i * m + j; // row i column j of C

                C[ic] += r * B[ib];
            }
        }
    }
}

/**
 * Print matrix M (stored row wise)
 */
void print_matrix(FP *M, const char *name, size_t nrow, size_t ncol)
{
    printf("Matrix %s\n", name);
    printf("[\n");
    for (size_t i = 0; i < nrow; i++)
    {
        printf("[");
        for (size_t j = 0; j < ncol; j++)
        {
            printf("%f,", M[i * ncol + j]);
        }
        printf("],\n");
    }
    printf("]\n");
}

int main(int argc, char **argv)
{

    FP *A, *B, *C, *HOST_C;    // matrices on host
    FP *dev_A, *dev_B, *dev_C; // matrices on device
    int n, p, m;               // matrix dimensions

    int gpucount = 0;  // Count of available GPUs
    int gpunum = 0;    // Device number to use
    int Grid_Dim_x = 1;  // Grid dimension, x and y
    int Grid_Dim_y = 1;
    int Block_Dim_x = 1; // Block dimension, x and y
    int Block_Dim_y = 1;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms;   // which is applicable for asynchronous code also
    hipError_t errorcode;

    // --------------------SET PARAMETERS AND DATA -----------------------
    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice)
    {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else
    {
        printf("Device count = %d\n", gpucount);
    }

    if (argc != 8)
    {
        printf("Usage: task1 <matrix dim n> <matrix dim p> <matrix dim m> <block dim x> <block dim y> <grid dim x> <grid dim y>\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);

    Block_Dim_x = atoi(argv[4]); // Rectangular block
    Block_Dim_y = atoi(argv[5]);
    if (Block_Dim_x * Block_Dim_y > 1024)
    {
        printf("Error, too many threads in block\n");
        exit(-1);
    }

    Grid_Dim_x = atoi(argv[6]);
    Grid_Dim_y = atoi(argv[7]);
    if (Grid_Dim_x * Block_Dim_x < m)
    {
        printf("%d,%d", Grid_Dim_x, Block_Dim_x);
        printf("Error, number of threads in x dimensions less than number of array elements\n");
        exit(-1);
    }
    if (Grid_Dim_y * Block_Dim_y < n)
    {
        printf("Error, number of threads in y dimensions less than number of array elements\n");
        exit(-1);
    }

    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    printf("Matrix Dimension = %d\n", n);
    printf("Block_Dim_x = %d, Block_Dim_y = %d, Grid_Dim_x = %d, Grid_Dim_y = %d\n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

    dim3 Grid(Grid_Dim_x, Grid_Dim_y);    // Grid structure
    dim3 Block(Block_Dim_x, Block_Dim_y); // Block structure

    size_t size_A = n * p * sizeof(FP);
    size_t size_B = p * m * sizeof(FP);
    size_t size_C = n * m * sizeof(FP);

    A = (FP *)malloc(size_A);      // dynamically allocated memory for arrays on host
    B = (FP *)malloc(size_B);      // dynamically allocated memory for arrays on host
    C = (FP *)malloc(size_C);      // results from GPU
    HOST_C = (FP *)malloc(size_C); // results from rectangular kij algorithm

    srand(12345);
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < p; j++)
        {
            A[i * p + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < p; i++)
    {
        for (size_t j = 0; j < n; j++)
        {
            B[i * n + j] = (FP)rand() / (FP)RAND_MAX;
        }
    }

    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < m; j++)
        {
            C[i * m + j] = 0.;
            HOST_C[i * m + j] = 0;
        }
    }

    // ------------- COMPUTATION DONE ON GPU ----------------------------
    hipMalloc((void **)&dev_A, size_A); // allocate memory on device
    hipMalloc((void **)&dev_B, size_B);
    hipMalloc((void **)&dev_C, size_C);
    hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    gpu_matrixmult_rectangular<<<Grid, Block>>>(dev_A, dev_B, dev_C, n, p, m);

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost);

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
    write_data_to_file("out/task1.csv", "task1a", "gpu", n, p, m, Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y,elapsed_time_ms);

    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // cudaEventRecord(start, 0); // use same timing

    // cpu_matrixmult_rectangular_kij(A, B, HOST_C, n, p, m); // do calculation on host

    // cudaEventRecord(stop, 0); // instrument code to measue end time
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&elapsed_time_ms, start, stop);

    // printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time
    // write_data_to_file("out/task1.csv", "task1a", "cpu", n, p, m, Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y,elapsed_time_ms);

    // ------------------- check device creates correct results -----------------
    // double error, suma, sumb, sumc, ai, bi, ci;
    // suma = 0.;
    // sumb = 0;
    // sumc = 0;
    // for (size_t i = 0; i < n * p; i++)
    // {
    //     ai = (double)A[i];
    //     suma += ai * ai;
    // }
    // for (size_t i = 0; i < p * m; i++)
    // {
    //     bi = (double)B[i];
    //     sumb += bi * bi;
    // }
    // for (size_t i = 0; i < n * m; i++)
    // {
    //     ci = (double)(C[i] - HOST_C[i]);
    //     sumc += ci * ci;
    // }
    // suma = sqrt(suma);
    // sumb = sqrt(sumb);
    // sumc = sqrt(sumc);
    // error = sumc / (suma * sumb);
    // printf("Approximate relative error between GPU and CPU: %e\n", error);

    // -------------- clean up ---------------------------------------
    free(A);
    free(B);
    free(C);
    free(HOST_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}